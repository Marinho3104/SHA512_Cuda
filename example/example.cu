#include "hip/hip_runtime.h"
/****************************** Marinho das Obras ******************************/
/****************************** Marinho das Obras ******************************/
/****************************** Marinho das Obras ******************************/
/****************************** Marinho das Obras ******************************/

#include "sha512.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void test() {

    sha512::cuda::Sha512_Context _ctx;

    sha512::cuda::Sha512_init( &_ctx );

    unsigned char _text[] = "abcdefghbcdefghicdefghijdefghijkefghijklfghijklmghijklmnhijklmnoijklmnopjklmnopqklmnopqrlmnopqrsmnopqrstnopqrstuabc";

    sha512::cuda::Sha512_hash( &_ctx, _text, 115 );

    unsigned char _hash_hex[ 129 ] = { 0 };

    sha512::cuda::Sha512_digest( &_ctx, _hash_hex );

    sha512::cuda::Sha512_free( &_ctx );

    printf("\n\n%s\n", _hash_hex );

}

int main() {

    test <<< 1, 1 >>>();

    hipDeviceSynchronize(); 

}

